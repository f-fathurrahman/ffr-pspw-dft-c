#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cuda/hipblas.h>

#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld)) + ((i)-1))

static __inline__ void modify(
    double *m, int ldm, int n, int p, int q, double alpha, double beta)
{
  printf("Calling hipblasDscal\n");
  printf("n-p+1 = %d\n",n-p+1);
  //hipblasDscal(n-p, alpha, &m[IDX2F(p,q,ldm)], ldm);
  hipblasDscal(ldm-p+1, beta, &m[IDX2F(p,q,ldm)], 1);
  printf("End calling hipblasDscal\n");
}


int main(int argc, char **argv)
{
  int i, j;
  hipblasStatus_t stat;
  double *d_A;
  double *A = 0;

  A = (double*)malloc(M*N*sizeof(*A));
  if(!A) {
    printf("Host memory allocation failed\n");
    return EXIT_FAILURE;
  }
  for(i=1; i<=M; i++) {
    for(j=1; j<=N; j++) {
      A[IDX2F(i,j,M)] = (double)((i-1)*M+j);
    }
  }

  printf("Matrix A before modify:\n");
  for(i=1; i<=M; i++) {
    for(j=1; j<=N; j++) {
      printf("%7.0f", A[IDX2F(i,j,M)]);
    }
    printf("\n");
  }

  cublasInit();
  printf("Allocating memory in GPU.\n");
  stat = cublasAlloc(M*N, sizeof(*A), (void**)&d_A);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf("Device memory allocation failed\n");
    cublasShutdown();
    return EXIT_FAILURE;
  }
  printf("Copy matrix to GPU.\n");
  stat = hipblasSetMatrix(M,N,sizeof(*A),A,M,d_A,M);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf("Data download failed.\n");
    cublasFree(d_A);
    cublasShutdown();
    return EXIT_FAILURE;
  }

  printf("Modifying matrix A\n");
  modify(d_A, M, N, 2, 3, 16.0, 12.0);
  printf("Copy matrix to CPU\n");
  stat = hipblasGetMatrix(M,N,sizeof(*A), d_A, M, A, M);
  if(stat != HIPBLAS_STATUS_SUCCESS) {
    printf("Data upload failed\n");
    cublasFree(d_A);
    cublasShutdown();
    return EXIT_FAILURE;
  }

  cublasFree(d_A);
  cublasShutdown();

  printf("Matrix A after modify:\n");
  for(i=1; i<=M; i++) {
    for(j=1; j<=N; j++) {
      printf("%7.0f", A[IDX2F(i,j,M)]);
    }
    printf("\n");
  }

  free(A);

  printf("Program ended normally\n");
  return EXIT_SUCCESS;
}

